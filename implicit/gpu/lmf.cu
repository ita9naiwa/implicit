#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>


#include "implicit/gpu/als.h"
#include "implicit/gpu/utils.cuh"

namespace implicit {

__global__ void lmf_update_kernel(int begin_index, int end_index,
                                  float* vec_deriv_sum,
                                  int samples,
                                  unsigned int * random_dislikes,
                                  int * indices, int * indptr, int * data,
                                  int factors,
                                  float * X, float * Y,
                                  float learning_rate, float reg,
                                  long neg_prop,
                                  int * stats) {
    extern __shared__ float shared_memory[];
    float * temp = &shared_memory[0];
    float deriv[factors];
    memset(deriv, 0, sizeof(float) * factors);
    int neg_processed = 0;

    for(int userid = begin_index + blockIdx.x; i < end_index; userid += gridDim.x) {
        int user_seen_item = indptr[userid + 1] - indptr[userid];
        if (user_seen_item == 0)
            continue;

        float * deriv_sum_sq = &vec_deriv_sum[userid * factors];
        // [indptr[i], indptr[i +1])까지가 user가 consume한 positive items;

        int likedid, dislikedid;

        float score;
        float * user = &X[userid * factors];
        for (int i=indptr[userid]; i < indptr[userid + 1]; ++i){
            likedid = indices[i];
            float * liked = &Y[likedid * factors];
            score = exp(dot(user, liked));
            float z = data[i] * (score/ (1.0 + score));
            float user_val = user[threadIdx.x],
                  liked_val = liked[threadIdx.x];
            deriv[threadIdx.x] += (data[i] - z) * liked[threadIdx.x]
        }

        for (int i = neg_prop * indptr[i]; neg_prop * indptr[j + 1]; ++j) {
            dislikedid = random_dislikes[i];
            float * dislikedid = &Y[dislikedid * factors];
            score = exp(dot(user, disliked));
            z = (score / (1.0 + score));
            deriv[threadIdx.x] -= z * disliked[threadIdx.x];
        }
        deriv[threadIdx.x] -= regularization * user[threadIdx.x];
        deriv_sum_sq[threadIdx.x] += deriv[threadIdx.x] * deriv[threadIdx.x];
        user[threadIdx.x] += (learning_rate * / (sqrt(1e-6 + deriv_sum_sq[threadIdx.x]))) * deriv[threadIdx.x];
    }
}

std::pair<int, int>  lmf_update(CudaDenseMatrix * vec_deriv_sum,
                                const CudaVector[int]& indices,
                                const CudaVector[int]& indptr,
                                const CudaVector[int]& data,
                                CudaDenseMatrix *X,
                                CudaDenseMatrix *Y,
                                float learning_rate,
                                float regularization,
                                long neg_prop,
                                long seed) {
    if (X->cols != Y->cols) throw std::invalid_argument("X and Y should have the same number of columns");
    if (userids.size != itemids.size)
        throw std::invalid_argument("userids and itemids should have same number of elements");
    // todo: check indptr = X->rows + 1

    int num_users = X.rows
    int nonzeros = userids.size;

    // allocate some memory
    int * stats;
    CHECK_CUDA(hipMalloc(&stats, sizeof(int) * 2));
    CHECK_CUDA(hipMemset(stats, 0, sizeof(int) * 2));

    // initialize memory for randomly picked positive/negative items
    unsigned int * random_dislikes;


    // Create a seeded RNG
    hiprandGenerator_t rng;
    CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));

    // Randomly pick values


    // TODO: multi-gpu support
    int devId;
    CHECK_CUDA(hipGetDevice(&devId));

    int multiprocessor_count;
    CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                      hipDeviceAttributeMultiprocessorCount,
                                      devId));

    int factors = X->cols;
    int block_count = 128 * multiprocessor_count;
    int thread_count = factors;
    int shared_memory_size = sizeof(float) * (factors);
    int begin_index = 0, end_index = 0;
    int num_samples = (1 << 16);
    CHECK_CUDA(hipMalloc(&random_dislikes, num_samples * neg_prop * sizeof(unsigned int)));
    // TODO: get rows passed in here
    while (end_index < num_users) {
        int num_positives = indptr[end_index] - indptr[begin_index];
        if (num_positives >= num_samples)
        {
            CHECK_CURAND(hiprandGenerate(rng, random_dislikes, num_samples * neg_prop));
            lmf_update_kernel<<<block_count, thread_count, shared_memory_size>>>(
                begin_index, end_index,
                deriv_sum_sq.data,
                nonzeros,
                random_dislikes,
                indices.data, indptr.data, data.data,
                factors,
                X->data, Y->data, learning_rate, reg,
                neg_prop,
                stats);
            begin_index = end_index;
            CHECK_CUDA(hipDeviceSynchronize());
        } else {
            end_index = += 1;
        }
    }
    if (begin_index != end_index) {
        CHECK_CURAND(hiprandGenerate(rng, random_dislikes, num_samples * neg_prop));
        lmf_update_kernel<<<block_count, thread_count, shared_memory_size>>>(
            begin_index, end_index,
            deriv_sum_sq.data,
            nonzeros,
            random_dislikes,
            indices.data, indptr.data, data.data,
            factors,
            X->data, Y->data, learning_rate, reg,
            neg_prop,
            stats);
    }

    // we're returning the number of correctly ranked items, get that value from the device
    int output[2];
    CHECK_CUDA(hipMemcpy(output, stats, 2 * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(random_dislikes));
    CHECK_CUDA(hipFree(stats));
    hiprandDestroyGenerator(rng);
    return std::make_pair(output[0], output[1]);
}
}
